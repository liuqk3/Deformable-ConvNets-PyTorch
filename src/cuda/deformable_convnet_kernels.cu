#include "hip/hip_runtime.h"
#include <cstdio>
#include "deformable_convnet_kernels.h"

#define CUDA_1D_KERNEL_LOOP(i, n) \
for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += blockDim.x * gridDim.x)

const int THREADS_PER_BLOCK = 1024; // the number of threads per block, i.e blockDim.x = 1024
inline int GET_NUM_BLOCKS(const int n)
{
    return (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
}


/**********************************************************************************************

The following codes are based on the code in: https://github.com/msracver/Deformable-ConvNets

**********************************************************************************************/
__device__ float deformable_im2col_bilinear(
    const float * bottom_data, // the input image starting from the top-left of filter
    const int data_width, // the width of input image
    const int height, // the rest height of image
    const int width, // the rest width of image
    float h, // the fraction h w.r.t. the top-left of filter
    float w // the fraction w w.r.t. the top-left of filter
){
    int h_low = floor(h);
    int w_low = floor(w);
    int h_high;
    int w_high;

    if (h_low >= height - 1)
    {
        h_high = h_low = height - 1;
        h = (float)h_low;
    }
    else
    {
        h_high = h_low + 1;
    }

    if (w_low >= width - 1)
    {
        w_high = w_low = width - 1;
        w = (float)w_low;
    }
    else
    {
        w_high = w_low + 1;
    }

    float top_left = bottom_data[h_low * data_width + w_low];
    float top_right = bottom_data[h_low * data_width + w_high];
    float bottom_left = bottom_data[h_high * data_width + w_low];
    float bottom_right = bottom_data[h_high * data_width + w_high];

    float lh = h - h_low;
    float lw = w - w_low;
    float hh = 1 - lh;
    float hw = 1 - lw;

    float w_tl = hw * hh, w_tr = lw * hh, w_bl = lh * hw, w_br = lh * lw;

    float val = w_tl * top_left + w_tr * top_right + w_bl * bottom_left + w_br * bottom_right;
    return val;
}


__device__ float get_im_weight( // get the weight of gradient w.r.t the data_im
    float argument_h, // fractional h
    float argument_w, // fractional w
    const int h, // the real h location in the image
    const int w, // the real w location in the image
    const int height, // the height of input image
    const int width // the width of input image
){
    if (argument_h < 0 || argument_h > height || argument_w < 0 || argument_w > width)
    {
        // empty
        return 0;
    }

    argument_h = max(argument_h, (float)0.0f);
    argument_w = max(argument_w, (float)0.0f);

    int argument_h_low = (int)argument_h;
    int argument_w_low = (int)argument_w;
    int argument_h_high;
    int argument_w_high;
    if (argument_h_low >= height - 1)
    {
        argument_h_high = argument_h_low = height - 1;
        argument_h = (float)argument_h_low;
    }
    else
    {
        argument_h_high = argument_h_low + 1;
    }

    if (argument_w_low >= width - 1)
    {
        argument_w_high = argument_w_low = width - 1;
        argument_w = (float)argument_w_low;
    }
    else
    {
        argument_w_high = argument_w_low + 1;
    }

    float weight = 0;
    if (h == argument_h_low)
    {
        if (w == argument_w_low)
        {
            weight = (h + 1 - argument_h) * (w + 1 - argument_w); // top_left
        }
        else if (w == argument_w_high)
        {
            weight = (h + 1 - argument_h) * (argument_w + 1 - w); // top_right
        }
    }
    else if (h == argument_h_high)
    {
        if (w == argument_w_low)
        {
            weight = (argument_h + 1 - h) * (w + 1 - argument_w); // bottom_left
        }
        else if (w == argument_w_high)
        {
            weight = (argument_h + 1 - h) * (argument_w + 1 - w); // bottom_right
        }
    }
    return weight;
}


__device__ float get_offset_weight( // get the weight of gradient w.r.t to the offset
    float argument_h, // the real fractional h in the image
    float argument_w, // the real fractional w in the image
    const int height, // the height of input image
    const int width, // the width of input image
    const float * im_data, // the input image data, starting current channel
    const int data_width, // the width of input image data
    const int bp_dir // denoting x or y
){
    if (argument_h < 0 || argument_h > height || argument_w < 0 || argument_w > width)
    {
        // empty
        return 0;
    }

    if (argument_h < 0)
        argument_h = 0;
    if (argument_w < 0)
        argument_w = 0;

    int argument_h_low = (int)argument_h;
    int argument_w_low = (int)argument_w;
    int argument_h_high;
    int argument_w_high;

    if (argument_h_low >= height - 1)
    {
        argument_h_high = argument_h_low = height - 1;
        argument_h = (float)argument_h_low;
    }
    else
    {
        argument_h_high = argument_h_low + 1;
    }
    if (argument_w_low >= width - 1)
    {
        argument_w_high = argument_w_low = width - 1;
        argument_w = (float)argument_w_low;
    }
    else
    {
        argument_w_high = argument_w_low + 1;
    }

    float weight = 0;
    if (bp_dir == 0) // coordinate h, or y
    {
        weight += -1 * (argument_w_low + 1 - argument_w) * im_data[argument_h_low * data_width + argument_w_low]; // top_left
        weight += -1 * (argument_w - argument_w_low) * im_data[argument_h_low * data_width + argument_w_high]; // top_right
        weight += (argument_w_low + 1 - argument_w) * im_data[argument_h_high * data_width + argument_w_low]; // bottom_left
        weight += (argument_w - argument_w_low) * im_data[argument_h_high * data_width + argument_w_high]; // bottom_right
    }
    else if (bp_dir == 1) // coordinate w, or x
    {
        weight += -1 * (argument_h_low + 1 - argument_h) * im_data[argument_h_low * data_width + argument_w_low]; // top_left
        weight += (argument_h_low + 1 - argument_h) * im_data[argument_h_low * data_width + argument_w_high]; // top_right
        weight += -1 * (argument_h - argument_h_low) * im_data[argument_h_high * data_width + argument_w_low]; // bottom_left
        weight += (argument_h - argument_h_low) * im_data[argument_h_high * data_width + argument_w_high]; // bottom_right
    }

    return weight;
}

__global__ void deformable_im2col_gpu_kernel(
    const int n,
    const float * data_im,
    const float * data_offset,
    const float * data_mask,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int strid_h,
    const int strid_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int height_col,
    const int width_col,
    float * data_col
){
    CUDA_1D_KERNEL_LOOP(index, n)
    {
        // index is the index of output matrix.
        // Note that the data_col is a 2D matrix,
        // but we can treat it as a 3D tensor with size
        // [input_channel * kernel_h * kernel_w, output_h, output_w]
        // then it can be indexed with the same mechanism that applied to
        // 3D tensors.
        const int w_col = index % width_col;
        const int h_col = (index / width_col) % height_col;
        const int c_im = (index / width_col) / height_col;
        const int c_col = c_im * kernel_h * kernel_w;

        // compute deformable group index
        const int deformable_group_index = c_im / channel_per_deformable_group;

        const int h_in = h_col * strid_h - pad_h;
        const int w_in = w_col * strid_w - pad_w; // (h_in, w_in) is the top_left coordinate of kernel in the data_im
        float * data_col_ptr = data_col + (c_col * height_col + h_col) * width_col + w_col;
        const float * data_im_ptr = data_im + (c_im * height + h_in) * width + w_in;
        const float * data_offset_ptr = // 2 is the x, y dimension
                data_offset + deformable_group_index * 2 * (kernel_h * kernel_w) * height_col * width_col;
        const float * data_mask_ptr = data_mask + deformable_group_index * (kernel_h * kernel_w) * height_col * width_col;

        for (int i = 0; i < kernel_h; ++i)
        {
            for (int j = 0; j < kernel_w; ++j)
            {
                const int data_offset_h_ptr = ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
                const int data_offset_w_ptr = ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col + w_col;
                const int data_mask_magnitude_ptr = ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;
                const float offset_h = data_offset_ptr[data_offset_h_ptr];
                const float offset_w = data_offset_ptr[data_offset_w_ptr];
                const float mask_magnitude = data_mask_ptr[data_mask_magnitude_ptr];

                float val = 0;

                // get the sampled coordinates
                const float h_im = h_in + i * dilation_h + offset_h;
                const float w_im = w_in + j * dilation_w + offset_w;
                if (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width)
                {
                    const float map_h = i * dilation_h + offset_h;
                    const float map_w = j * dilation_w + offset_w;
                    const int cur_height = height - h_in; // the rest height
                    const int cur_width = width - w_in; // the rest width

                    val = deformable_im2col_bilinear(data_im_ptr, width, cur_height, cur_width, map_h, map_w);
                }

                * data_col_ptr = val * mask_magnitude;
                data_col_ptr += height_col * width_col; // change to the next row in 2D matrix data_col
            }
        }
    }
}


// https://blog.csdn.net/mrhiuser/article/details/52672824?tdsourcetag=s_pctim_aiomsg
void deformable_im2col(
    hipStream_t stream,
    const float * data_im,
    const float * data_offset,
    const float * data_mask,
    const int channels,
    const int height, // the height of input data_im
    const int width, // the width of input data_im
    const int ksize_h,
    const int ksize_w,
    const int pad_h,
    const int pad_w,
    const int strid_h,
    const int strid_w,
    const int dilation_h,
    const int dilation_w,
    const int deformable_group, // the feature maps are divided into deformable_group groups along
                                // channel dimension, the features in each group share the same offsets.
                                // And it is set to 1 in the deformable paper, that is all features share
                                // the same offsets.
    float * data_col // the data changed to column, [input_channel * k_h * k_w, out_height * out_width]
){
    // we are going to launch channels * height_col * width_col threads,
    // each thread responsible for copying a sub-column with length ksize_h * ksize_w
    // in 2D matrix data_col.
    int height_col = (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / strid_h + 1;
    int width_col = (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / strid_w + 1;
    int num_threads = channels * height_col * width_col;
    int channel_per_deformable_group = channels / deformable_group;

    // launch
    int num_blocks = GET_NUM_BLOCKS(num_threads);
    deformable_im2col_gpu_kernel<<<num_blocks, THREADS_PER_BLOCK, 0, stream>>>(
        num_threads,
        data_im,
        data_offset,
        data_mask,
        height,
        width,
        ksize_h,
        ksize_w,
        pad_h,
        pad_w,
        strid_h,
        strid_w,
        dilation_h,
        dilation_w,
        channel_per_deformable_group,
        height_col,
        width_col,
        data_col);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("\n error in deformable_im2col: %s\n", hipGetErrorString(err));
    }
}


__global__ void deformable_col2im_gpu_kernel(
    const int n,
    const float * data_col,
    const float * data_offset,
    const float * data_mask,
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int height_col,
    const int width_col,
    float * grad_im
){
    CUDA_1D_KERNEL_LOOP(index, n) // index is the index of data_col
    {
        // i, j is the index in the filter (weight of this conv)
        const int j = (index / width_col / height_col) % kernel_w;
        const int i = (index / width_col / height_col / kernel_w) % kernel_h;

        const int c_im = index / width_col / height_col / kernel_w / kernel_h; // the channel of input data_im
        // compute the start and end of the output

        const int deformable_group_index = c_im / channel_per_deformable_group;

        int w_col = index % width_col;
        int h_col = (index / width_col) % height_col;
        int w_in = w_col * stride_w - pad_w;
        int h_in = h_col * stride_h - pad_h; // w_in, h_in is the coordinate of top_left of this kernel in the input data

        const float * data_offset_ptr = data_offset + deformable_group_index * 2 * kernel_h * kernel_w * height_col * width_col;
        const int data_offset_h_ptr = ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr = ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col + w_col;
        const float offset_h = data_offset_ptr[data_offset_h_ptr];
        const float offset_w = data_offset_ptr[data_offset_w_ptr];
        const float cur_inv_h_data = h_in + i * dilation_h + offset_h;
        const float cur_inv_w_data = w_in + j * dilation_w + offset_w;

        const float * data_mask_ptr = data_mask + deformable_group_index * kernel_h * kernel_w * height_col * width_col;
        const int data_mask_magnitude_ptr = ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;
        const float mask_magnitude = data_mask_ptr[data_mask_magnitude_ptr];

        const float cur_top_grad = data_col[index];
        const int cur_h = (int)cur_inv_h_data;
        const int cur_w = (int)cur_inv_w_data;

        // TODO: simplify following back propagation
        for (int dy = -2; dy <= 2; dy++)
        {
            if (cur_h + dy >= 0 && cur_h + dy < height && abs(cur_inv_h_data - (cur_h + dy)) < 1)
            {
                for (int dx = -2; dx <= 2; dx++)
                {
                    if (cur_w + dx >= 0 && cur_w + dx < width && abs(cur_inv_w_data - (cur_w + dx)) < 1)
                    {
                        int cur_bottom_grad_pos = (c_im * height + cur_h + dy) * width + cur_w + dx;
                        float weight = get_im_weight(
                            cur_inv_h_data,
                            cur_inv_w_data,
                            cur_h + dy,
                            cur_w + dx,
                            height,
                            width);
                        atomicAdd(grad_im + cur_bottom_grad_pos, weight * cur_top_grad * mask_magnitude);
                    }
                }
            }
        }
    }
}


void deformable_col2im( // backward to get the grads w.r.t input feature map
    hipStream_t stream,
    const float * data_col, // the grads w.r.t data_col
    const float * data_offset,
    const float * data_mask,
    const int channels,
    const int height,
    const int width,
    const int ksize_h,
    const int ksize_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int deformable_group, // the feature maps are divided into deformable_group groups along
                                // channel dimension, the features in each group share the same offsets.
                                // And it is set to 1 in the deformable paper, that is all features share
                                // the same offsets.
    float * grad_im
){
    int height_col = (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
    int width_col = (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
    int num_threads = channels * ksize_h * ksize_w * height_col * width_col;
    int channel_per_deformable_group = channels / deformable_group;

    // to avoid involving atomic operations, we will lauch one thread per bottom dimension,
    // and then in the kernel add up the top dimensions.
    int num_blocks = GET_NUM_BLOCKS(num_threads);
    deformable_col2im_gpu_kernel<<<num_blocks, THREADS_PER_BLOCK, 0, stream>>>(
        num_threads,
        data_col,
        data_offset,
        data_mask,
        channels,
        height,
        width,
        ksize_h,
        ksize_w,
        pad_h,
        pad_w,
        stride_h,
        stride_w,
        dilation_h,
        dilation_w,
        channel_per_deformable_group,
        height_col,
        width_col,
        grad_im);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("\n error in deformable_col2im: %s", hipGetErrorString(err));
    }
}


__global__ void deformable_col2im_offset_gpu_kernel(
    const int n, // number of threads
    const float * data_col,
    const float * data_im,
    const float * data_offset,
    const float * data_mask,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int height_col,
    const int width_col,
    float * grad_offset)
{
    CUDA_1D_KERNEL_LOOP(index, n) // index is the index of grad_offset
    {
        float val = 0;

        // compute the start of the output
        // w and h are the index of data_col, while c is the index of data_offset
        int w_col = index % width_col;
        int h_col = (index / width_col) % height_col;
        int c = index / width_col / height_col;

        const int deformable_group_index = c / (2 * kernel_h * kernel_w); // if deformable_group is 1, this value is 0
                                                                          // deformable_group = c / width_col / height_col
                                                                          // c is the number of channels of offset
        const int col_step = kernel_h * kernel_w;
        int cnt = 0;
        const float * data_col_ptr = // treat data_col as a 3D tensor with size [c, h, w], where c = input_c * k_h * k_w
            data_col + deformable_group_index * channel_per_deformable_group * width_col * height_col;
        const float * data_im_ptr =
            data_im + deformable_group_index * channel_per_deformable_group / kernel_h / kernel_w * height * width;
        const float * data_offset_ptr =
            data_offset + deformable_group_index * 2 * kernel_h * kernel_w * height_col * width_col;
        const float * data_mask_ptr =
            data_mask + deformable_group_index * kernel_h * kernel_w * height_col * width_col;
        int offset_c = c - deformable_group_index * 2 * kernel_h * kernel_w;

        for (int col_c = (offset_c / 2); col_c < channel_per_deformable_group; col_c += col_step)
        {   // each group, the first k_h * k_w channels are y offsets, and the rest k_h * k_w channels are x offsets
            const int col_pos = ((col_c * height_col) + h_col) * width_col + w_col;
            const int bp_dir = offset_c % 2; // compute y (0) or w (1)

            int j = (col_pos / width_col / height_col) % kernel_w;
            int i = ((col_pos / width_col / height_col) / kernel_w) % kernel_h;
            int w_in = w_col * stride_w - pad_w;
            int h_in = h_col * stride_h - pad_h;

            const int data_offset_h_ptr = (((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col);
            const int data_offset_w_ptr = (((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col + w_col);
            const float offset_h = data_offset_ptr[data_offset_h_ptr];
            const float offset_w = data_offset_ptr[data_offset_w_ptr];
            float inv_h = h_in + i * dilation_h + offset_h;
            float inv_w = w_in + i * dilation_w + offset_w;

            const int data_mask_magnitude_ptr = ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;
            const float mask_magnitude = data_mask_ptr[data_mask_magnitude_ptr];

            if (inv_h < 0 || inv_w < 0 || inv_h >= height || inv_w >= width)
            {
                inv_h = inv_w = -1;
            }

            const float weight = get_offset_weight(
                inv_h,
                inv_w,
                height,
                width,
                data_im_ptr + cnt * height * width,
                width,
                bp_dir);
            val += weight * data_col_ptr[col_pos] * mask_magnitude;
            cnt += 1;
        }

        grad_offset[index] = val;
    }
}


void deformable_col2im_offset( // backward to get the grads w.r.t the offset
    hipStream_t stream,
    const float * data_col, // the grad w.r.t the data_col
    const float * data_im,
    const float * data_offset,
    const float * data_mask,
    const int channels,
    const int height,
    const int width,
    const int ksize_h,
    const int ksize_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int deformable_group,
    float * grad_offset
){
    int height_col = (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
    int width_col = (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;

    // to avoid involving atomic operations, we will launch one thread per
    // bottom dimension, and then in the thread add up the top dimensions.
    int num_threads = height_col * width_col * 2 * ksize_h * ksize_w * deformable_group;

    int channel_per_deformable_group = channels * ksize_h * ksize_w / deformable_group;

    int num_blocks = GET_NUM_BLOCKS(num_threads);
    deformable_col2im_offset_gpu_kernel<<<num_blocks, THREADS_PER_BLOCK, 0, stream>>>(
        num_threads,
        data_col,
        data_im,
        data_offset,
        data_mask,
        height,
        width,
        ksize_h,
        ksize_w,
        pad_h,
        pad_w,
        stride_h,
        stride_w,
        dilation_h,
        dilation_w,
        channel_per_deformable_group,
        height_col,
        width_col,
        grad_offset);

     hipError_t err = hipGetLastError();
     if (err != hipSuccess)
     {
        printf("\n error in deformable_col2im_offset: %s\n ", hipGetErrorString(err));
     }
}


__global__ void deformable_col2im_mask_gpu_kernel(
    const int n, // number of threads
    const float * data_col,
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int deformable_group,
    const int height_col,
    const int width_col,
    float * grad_mask)
{
    CUDA_1D_KERNEL_LOOP(index, n) // index is the index of grad_mask
    {
        float val = 0;

        // note that:
        // num_threads = deformable_group * (ksize_h * ksize_w) * height_col * width_col;
        // so, index = (((group_index * (ksize_h + i) * ksize_w + j) * height_col + h_col) * width_col + w_col;

        // compute the start of the output
        // w_col and h_col are the index of data_col, while c is the index of data_mask
        int w_col = index % width_col;
        int h_col = (index / width_col) % height_col;
        int c = (index / width_col / height_col);

        const int deformable_group_index = c / (kernel_h * kernel_w);

        const int col_step = kernel_h * kernel_w;
        const int channel_per_deformable_group = channels * kernel_h * kernel_w / deformable_group;

        const float * data_col_ptr =
            data_col + deformable_group_index * channel_per_deformable_group * width_col * height_col;
        int mask_c = c - deformable_group_index * (kernel_h * kernel_w);
        for (int col_c = mask_c; col_c < channel_per_deformable_group; col_c += col_step)
        {
            int col_pos = (col_c * height_col + h_col) * width_col + w_col;
            val += data_col_ptr[col_pos];
        }

        grad_mask[index] = val;

    }
}

void deformable_col2im_mask(
    hipStream_t stream,
    const float * data_col, // the grad w.r.t the data_col
    const int channels,
    const int height,
    const int width,
    const int ksize_h,
    const int ksize_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int deformable_group,
    float * grad_mask
){
    int height_col = (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
    int width_col = (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;

    // to avoid involving atomic operations, we will launch one thread per
    // bottom dimension, and then in the thread add up the top dimensions.
    int num_threads = deformable_group * (ksize_h * ksize_w) * height_col * width_col;

    // printf("deformable_col2im_mask, num_threads: %d \n", num_threads);
    int num_blocks = GET_NUM_BLOCKS(num_threads);

    deformable_col2im_mask_gpu_kernel<<<num_blocks, THREADS_PER_BLOCK, 0, stream>>>(
        num_threads,
        data_col,
        channels,
        height,
        width,
        ksize_h,
        ksize_w,
        deformable_group,
        height_col,
        width_col,
        grad_mask);

    //printf("deformable_col2im_mask, num_threads: %d \n", num_threads);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
       printf("\n error in deformable_col2im_mask: %s\n", hipGetErrorString(err));
    }
}


/**********************************************************************************************

The following codes are based on the code in: https://github.com/longcw/RoIAlign.pytorch

**********************************************************************************************/

























